#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix.h"
#include "stdio.h"
#include "stdlib.h"

__global__ void kernUpSweep(int n, int d, const float *idata, float *odata) {
  if (IDx >= n) return;
  int addTerm = (IDx + 1) % (d * 2) == 0 ? idata[IDx - d] : 0;
  odata[IDx] = idata[IDx] + addTerm;
}

__global__ void kernDownSweep(int length, int d, const float *idata, float *odata) {
  if (IDx >= length) return;

  // On the first iteration, and using only one thread, set the last element to 0.
  if ((IDx + 1) % d == 0) {
    int swapIndex = IDx - (d / 2);
    int term = (length == d) && (IDx == d - 1) ? 0 : idata[IDx];
    odata[IDx] = term + idata[swapIndex];
    odata[swapIndex] = term;
  }
}

void dev_scan(int n, const float *dev_idata, float *dev_odata) {

  // round n up to the nearest power of 2
  int bufferedLength = pow(2, ceil(log2((float)n))); 
  dim3 numBlocks = blockcount(bufferedLength); // enough blocks to allocate one thread to each array element

  float *dev_temp;
  hipError_t cudaStat = hipMalloc((void**)&dev_temp,
      bufferedLength*sizeof(*dev_temp));
  check(cudaStat != hipSuccess, "hipMalloc failed for `temp` in `reduce_avg`");
  hipMemcpy(dev_temp, dev_idata, n * sizeof(*dev_temp),
      hipMemcpyDeviceToDevice);

  // upsweep
  for (int d = 1; d <= n; d *= 2) {
    kernUpSweep <<<numBlocks, BLOCKSIZE >>>(n, d, dev_temp, dev_odata);
    std::swap(dev_temp, dev_odata);
  }

  // downsweep
  for (int d = bufferedLength; d >= 1; d /= 2) {
    kernDownSweep << <numBlocks, BLOCKSIZE >> >(bufferedLength, d, dev_temp, dev_odata);
    std::swap(dev_temp, dev_odata);
  }
}
