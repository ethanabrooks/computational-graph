#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "util.h" 

#define MAT_IDX(i, j, width) ((i) * (width) + (j))

hipblasHandle_t handle = NULL;

extern "C" {
  void maybe_init_cublas() {
    hipblasStatus_t stat = hipblasCreate(&handle);
    check(stat != HIPBLAS_STATUS_SUCCESS, "CUBLAS initialization failed"); 
  };

  int size(const Matrix *m) { return m->width * m->height; }

  void download_matrix(const Matrix *src, float *dst) {
    hipblasStatus_t stat = hipblasGetMatrix(src->width, src->height, 
        sizeof(*src->array), 
        src->array, src->width, dst, src->width);
    check(stat != HIPBLAS_STATUS_SUCCESS, "download_matrix failed");
  }


  __global__ 
  void _transpose(int len, float *out, const float *in, int lda, int ldb) {
    SET(out, in[(IDx % lda) * ldb + int(IDx / lda)])
  }

  void upload_matrix(const float *src, Matrix *dst) {
    float *temp = safe_cuda_malloc<float>(size(dst));
    host2device<float>(size(dst), src, temp);
    DEFAULT_LAUNCH(_transpose, dst, temp, dst->height, dst->width);
    hipFree(temp);
  }

  // allocates on device
  void alloc_matrix(Matrix *matrix, int height, int width) { 
    if (!handle) {
      hipblasStatus_t stat = hipblasCreate(&handle);
      check(stat != HIPBLAS_STATUS_SUCCESS, "CUBLAS initialization failed"); 
    }
    matrix->width = width;
    matrix->height = height;

    // allocate space for matrix on GPU 
    hipError_t cudaStat = hipMalloc((void**)&matrix->array, 
        width*height*sizeof(*matrix->array)); 
    check(cudaStat != hipSuccess, "device memory allocation failed"); 
  }

  void init_matrix(Matrix *matrix, const float *array, int height, int width) {
    alloc_matrix(matrix, height, width);
    upload_matrix(array, matrix);
  }

  void copy_matrix(const Matrix *src, Matrix *dst) {
    dst->height = src->height;
    dst->width = src->width;

    hipError_t stat = device2device<float>(size(src), 
        src->array, dst->array);
    check(stat != hipSuccess, "copy_matrix failed");
  }

  __global__ 
  void _memset(int len, float *array, float value) {
    SET(array, value);
  }

  void fill_matrix(Matrix *matrix, float value) {
    DEFAULT_LAUNCH(_memset, matrix, value);
  }


  void print_matrix(const Matrix *matrix) {
  // allocate space for matrix on CPU 
  float *array = safe_malloc<float>(size(matrix));

  // copy matrix to CPU
  download_matrix(matrix, array);

  int i, j;
  rng(i, 0, matrix->height) {
    rng(j, 0, matrix->width) {
      printf("%7.0f", array[MAT_IDX(j, i, matrix->height)]);
    }
    printf("\n");
  }

    free(array);
  }

  void free_matrix(Matrix *matrix) {
    hipFree(matrix->array);
  }
}

