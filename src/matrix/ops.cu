#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "matrix.h"
#include "util.h"

#define IDx_T ((IDx) % (width)) * (width) + ((IDx) / (width))
#define UN_MAP(name, f_body) \
  __device__ \
  float f_ ## name(float x) { \
    return f_body; \
  } \
  __global__ \
  void _ ## name(int len, float *result, const float *a) { \
    SET(result, f_ ## name(a[IDx])) \
  } \
  void map_ ## name(const Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name, result, m->dev_array); \
  }

#define BIN_BROADCAST(name, op) \
  __global__ \
  void _ ## name ## _scalar(int len, float *result, const float *a, float val) { \
    SET(result, val op a[IDx]) \
  } \
  void broadcast_ ## name(float val, const Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar, result, m->dev_array, val); \
  }

#define BIN_BROADCAST_REV(name, op) \
  __global__ \
  void _ ## name ## _scalar_rev(int len, float *result, const float *a, float val) { \
    SET(result, a[IDx] op val) \
  } \
  void broadcast_ ## name ## _rev(const Matrix *m, float val, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar_rev, result, m->dev_array, val); \
  }

#define BIN_ELEMWISE(name, op) \
  __global__ \
  void _ ## name (int len, float *result, const float *a1, const float *a2) { \
    SET(result, a1[IDx] op a2[IDx]) \
  } \
  void elemwise_ ## name (const Matrix *m1, const Matrix *m2, Matrix *result) { \
    check_all_eq(m1, m2, result); \
    DEFAULT_LAUNCH(_ ## name, result, m1->dev_array, m2->dev_array); \
  }
#define CHECK_EQUAL(side1, side2) \
  check(side1 != side2,  #side1 " must equal " #side2)

void check_all_eq(const Matrix *m1, const Matrix *m2, const Matrix *result) {
  CHECK_EQUAL(m1->height, m2->height);
  CHECK_EQUAL(m1->width, m2->width);
  CHECK_EQUAL(m1->height, result->height);
  CHECK_EQUAL(m1->width, result->width);
}

extern "C" {
  UN_MAP(neg, -x) // map_neg
  UN_MAP(sq, x * x) // map_sq
  UN_MAP(abs, x < 0 ? -x : x) // map_aps
  UN_MAP(signum, x < 0 ? -1 : 1) // map_signum
  UN_MAP(sigmoid, 1.0f / (1.0f + expf(-x))) // map_sigmoid
  UN_MAP(tanh, tanh(x)) // map_tanh
  UN_MAP(one_minus, 1.0f - x) // map_one_minus

  BIN_ELEMWISE(mul, *) // elemwise_mult
  BIN_ELEMWISE(add, +) // elemwise_add
  BIN_ELEMWISE(sub, -) // elemwise_sub

  BIN_BROADCAST(mul, *) // broadcast_mult
  BIN_BROADCAST(add, +) // broadcast_add
  BIN_BROADCAST(sub, -) // broadcast_sub

  BIN_BROADCAST_REV(sub, -) // broadcast_sub_rev
  BIN_BROADCAST_REV(mul, *) // broadcast_mul_rev
  BIN_BROADCAST_REV(add, +) // broadcast_add_rev

  void gemm(const Matrix *m1, bool trans1,
            const Matrix *m2, bool trans2,
            Matrix *result) {

    if (trans1) {
      CHECK_EQUAL(m1->width, result->height);
      if (trans2) {
        CHECK_EQUAL(m1->height, m2->width);
        CHECK_EQUAL(m2->height, result->width);
      } else {
        CHECK_EQUAL(m1->height, m2->height);
        CHECK_EQUAL(m2->width, result->width);
      }
    } else {
      CHECK_EQUAL(m1->height, result->height);
      if (trans2) {
        CHECK_EQUAL(m1->width, m2->width);
        CHECK_EQUAL(m2->height, result->width);
      } else {
        CHECK_EQUAL(m1->width, m2->height);
        CHECK_EQUAL(m2->width, result->width);
      }
    }

    float alpha = 1;
    float beta = 0;
    hipblasStatus_t stat = hipblasSgemm(handle,
        trans1 ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        trans2 ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        result->height,     // m
        result->width,      // n
        trans1 ? m1->height : m1->width,
        &alpha,             // alpha
        m1->dev_array,      // A
        m1->height,         // lda
        m2->dev_array,      // B
        m2->height,         // ldb
        &beta,              // beta
        result->dev_array,  // C
        result->height);    // ldc
    switch (stat) {
      case HIPBLAS_STATUS_NOT_INITIALIZED:
        fprintf(stderr,
            "GEMM failed. Cublas not initialized.\n");
        break;
      case HIPBLAS_STATUS_INVALID_VALUE:
        fprintf(stderr,
            "GEMM failed. Invalid value.\n");
        break;
      case HIPBLAS_STATUS_ARCH_MISMATCH:
        fprintf(stderr,
            "GEMM failed. The device does not support the operation.\n");
        break;
      case HIPBLAS_STATUS_EXECUTION_FAILED:
        fprintf(stderr,
            "GEMM failed. The function failed to launch on the GPU.\n");
        break;
    }
    check(stat != HIPBLAS_STATUS_SUCCESS, "gemm failed :(");
  }

  __global__
  void _reduce_equal(int len, const float *a, unsigned int *boolean, float x) {
    if (IDx >= len) return;
    atomicAnd(boolean, a[IDx] == x);
  }

  __global__
  void _reduce_sum(int len, const float *a, float *sum) {
    if (IDx >= len) return;
    atomicAdd(sum, a[IDx]);
  }

  bool reduce_equal(const Matrix *m, float x) {
    unsigned int *dev_bool = safe_cuda_malloc<unsigned int>(1);
    unsigned int t = 1;

    hipError_t cudaStat = host2device(1, &t, dev_bool);
    check(cudaStat != hipSuccess, "host2device failed in reduce_eq");

    _reduce_equal<<<blockcount(size(m)), BLOCKSIZE>>>
      (size(m), m->dev_array, dev_bool, x);

    cudaStat = device2host(1, dev_bool, &t);
    check(cudaStat != hipSuccess, "device2host failed in reduce_sum");

    hipFree(dev_bool);
    return t == 1;
  }

  float reduce_sum(const Matrix *m) {
    float *dev_sum = safe_cuda_malloc<float>(1);
    float sum = 0;

    hipError_t cudaStat = host2device(1, &sum, dev_sum);
    check(cudaStat != hipSuccess, "host2device failed in reduce_sum");

    _reduce_sum<<<blockcount(size(m)), BLOCKSIZE>>>
      (size(m), m->dev_array, dev_sum);

    cudaStat = device2host(1, dev_sum, &sum);
    check(cudaStat != hipSuccess, "device2host failed in reduce_sum");

    hipFree(dev_sum);
    return sum;
  }
}

