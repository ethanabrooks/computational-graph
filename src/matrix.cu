#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "util.h" 

extern "C" {
  __global__ 
  void _memset(int len, float *array, float value) {
    SET(array, value);
  }

  int size(const Matrix *m) { return m->width * m->height; }

  void copy_dev2dev(Matrix *src, Matrix *dst) {
      hipError_t stat = hipMemcpy(dst->dev_array, src->dev_array, 
          src->height * src->width * sizeof(*src->dev_array),
          hipMemcpyDeviceToDevice);
      check(stat != hipSuccess, "copy_dev2dev failed");
  }

  void download_matrix(const Matrix *src, float *dst) {
    hipblasStatus_t stat = hipblasGetMatrix(src->width, src->height, 
        sizeof(*src->dev_array), 
        src->dev_array, src->width, dst, src->width);
    check(stat != HIPBLAS_STATUS_SUCCESS, "download_matrix failed");
  }

  void upload_matrix(float *src, const Matrix *dst) {
    hipblasStatus_t blas_stat = hipblasSetMatrix(dst->width, dst->height, 
        sizeof(*src), src, dst->width, dst->dev_array, dst->width); 
    check(blas_stat != HIPBLAS_STATUS_SUCCESS, "upload_matrix failed"); 

    /*hipError_t custat = hipMemcpy(src, dst->dev_array,*/
        /*size(dst) * sizeof(*src), hipMemcpyDeviceToHost);*/
    /*check(custat != hipSuccess, "data upload failed"); */
  }

  // allocates on device
  void alloc_matrix(Matrix *matrix, int height, int width) { 
    matrix->width = width;
    matrix->height = height;

    // allocate space for matrix on GPU 
    hipError_t cudaStat = hipMalloc((void**)&matrix->dev_array, 
        width*height*sizeof(*matrix->dev_array)); 
    check(cudaStat != hipSuccess, "device memory allocation failed"); 
  }

  void init_matrix(Matrix *matrix, float *array, int height, int width) {
    alloc_matrix(matrix, height, width);
    upload_matrix(array, matrix);
  }

  void copy_matrix(Matrix *src, Matrix *dst) {
  }

  void fill_matrix(Matrix *matrix, float value) {
    DEFAULT_LAUNCH(_memset, matrix, value)
  }

  void print_matrix(Matrix *matrix) {

    // allocate space for matrix on CPU 
    float *array = safe_malloc<float>(size(matrix));

    // copy matrix to CPU
    download_matrix(matrix, array);

    int i, j;
    rng(j, 0, matrix->height) {
      rng(i, 0, matrix->width) {
        printf("%7.0f", array[idx2c(i, j, matrix->width)]);
      }
      printf("\n");
    }

    free(array);
  }

  void free_matrix(Matrix *matrix) {
    hipFree(matrix->dev_array);
  }
}

