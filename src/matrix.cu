#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "util.h" 

hipblasHandle_t handle;

extern "C" {

  void init_cublas() {
    hipblasStatus_t stat = hipblasCreate(&handle);
    check(stat != HIPBLAS_STATUS_SUCCESS, "CUBLAS initialization failed"); 
  }

  int size(const Matrix *m) { return m->width * m->height; }

  void download_matrix(const Matrix *src, float *dst) {
    hipblasStatus_t stat = hipblasGetMatrix(src->width, src->height, 
        sizeof(*src->dev_array), 
        src->dev_array, src->width, dst, src->width);
    check(stat != HIPBLAS_STATUS_SUCCESS, "download_matrix failed");
  }

  void upload_matrix(const float *src, Matrix *dst) {
    hipblasStatus_t blas_stat = hipblasSetMatrix(dst->width, dst->height, 
        sizeof(*src), src, dst->width, dst->dev_array, dst->width); 
    check(blas_stat != HIPBLAS_STATUS_SUCCESS, "upload_matrix failed"); 
  }

  // allocates on device
  void alloc_matrix(Matrix *matrix, int height, int width) { 
    matrix->width = width;
    matrix->height = height;

    // allocate space for matrix on GPU 
    hipError_t cudaStat = hipMalloc((void**)&matrix->dev_array, 
        width*height*sizeof(*matrix->dev_array)); 
    check(cudaStat != hipSuccess, "device memory allocation failed"); 
  }

  void init_matrix(Matrix *matrix, const float *array, int height, int width) {
    alloc_matrix(matrix, height, width);
    upload_matrix(array, matrix);
  }

  void copy_matrix(const Matrix *src, Matrix *dst) {
    dst->height = src->height;
    dst->width = src->width;
    hipError_t stat = device2device<float>(size(src), 
        src->dev_array, dst->dev_array);
    check(stat != hipSuccess, "copy_matrix failed");
  }

  __global__ 
  void _memset(int len, float *array, float value) {
    SET(array, value);
  }

  void fill_matrix(Matrix *matrix, float value) {
    DEFAULT_LAUNCH(_memset, matrix, value)
  }

  void print_matrix(const Matrix *matrix) {

    // allocate space for matrix on CPU 
    float *array = safe_malloc<float>(size(matrix));

    // copy matrix to CPU
    download_matrix(matrix, array);

    int i, j;
    rng(j, 0, matrix->height) {
      rng(i, 0, matrix->width) {
        printf("%7.0f", array[idx2c(i, j, matrix->width)]);
      }
      printf("\n");
    }

    free(array);
  }

  void free_matrix(Matrix *matrix) {
    hipFree(matrix->dev_array);
  }
}

