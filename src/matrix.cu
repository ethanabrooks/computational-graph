#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 

int double_input(int input) {
  return 2 * input;
}

dim3 blockcount(int count) {
  return dim3(count / BLOCKSIZE.x + 1);
}

void check(int condition, const char *msg) {
  if (condition) {
    fprintf(stderr, "%s\n", msg);
    exit(EXIT_FAILURE);
  }
}

int size(Matrix m) { return m.width * m.height; }

__global__ 
void _memset(int len, float *array, float value) {
  SET(array, value);
}

void alloc_matrix(Matrix *matrix, int height, int width) {
  matrix->width = width;
  matrix->height = height;

  // allocate space for matrix on GPU 
  hipError_t cudaStat = hipMalloc((void**)&matrix->devArray, 
      width*height*sizeof(*matrix->devArray)); 
  check(cudaStat != hipSuccess, "device memory allocation failed"); 
}

void init_matrix(Matrix *matrix, float *array, int height, int width) {
  alloc_matrix(matrix, height, width);

  // copy matrix to GPU 
  hipblasStatus_t stat = hipblasSetMatrix(width, height, sizeof(*array), 
      array, width, matrix->devArray, width); 
  check(stat != HIPBLAS_STATUS_SUCCESS, "data upload failed"); 

  hipMemcpy(array, matrix->devArray, height * width * sizeof(*array), hipMemcpyDeviceToHost);
}

void copy_matrix(Matrix *src, Matrix *dst) {
  alloc_matrix(dst, src->height, src->width);

  // copy matrix from src
  hipMemcpy(dst->devArray, src->devArray, 
      src->height * src->width * sizeof(*src->devArray), hipMemcpyDeviceToDevice);
}

void fill_matrix(Matrix *matrix, float value) {
  DEFAULT_LAUNCH(_memset, matrix, value);
}

void print_matrix(Matrix *matrix) {
  hipblasStatus_t stat;

  // allocate space for matrix on CPU 
  float *array = (float *)malloc(matrix->width * matrix->height *
      sizeof(*matrix->devArray)); 
  check(!array, "host memory allocation failed"); 


  // copy matrix to CPU
  stat = hipblasGetMatrix(matrix->width, matrix->height, sizeof(*matrix->devArray), 
      matrix->devArray, matrix->width, array, matrix->width);
  check(stat != HIPBLAS_STATUS_SUCCESS, "data download failed");

  int i, j;
  range(j, 0, matrix->height) {
    range(i, 0, matrix->width) {
      printf("%7.0f", array[idx2c(i, j, matrix->width)]);
    }
    printf("\n");
  }
}

