#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "util.h" 

extern "C" {
  __global__ 
  void _memset(int len, float *array, float value) {
    SET(array, value);
  }

  int size(const Matrix *m) { return m->width * m->height; }

  // allocates on device
  void alloc_matrix(Matrix *matrix, int height, int width) { 
    matrix->width = width;
    matrix->height = height;

    // allocate space for matrix on GPU 
    hipError_t cudaStat = hipMalloc((void**)&matrix->dev_array, 
        width*height*sizeof(*matrix->dev_array)); 
    check(cudaStat != hipSuccess, "device memory allocation failed"); 
  }

  void init_matrix(Matrix *matrix, float *array, int height, int width) {
    alloc_matrix(matrix, height, width);

    // copy matrix to GPU 
    hipblasStatus_t stat = hipblasSetMatrix(width, height, sizeof(*array), 
        array, width, matrix->dev_array, width); 
    check(stat != HIPBLAS_STATUS_SUCCESS, "data upload failed"); 

    hipMemcpy(array, matrix->dev_array,
        height * width * sizeof(*array),
        hipMemcpyDeviceToHost);
  }

  void copy_matrix(Matrix *src, Matrix *dst) {
     
    // copy matrix from src
    hipMemcpy(dst->dev_array, src->dev_array, 
        src->height * src->width * sizeof(*src->dev_array),
        hipMemcpyDeviceToDevice);
  }

  void fill_matrix(Matrix *matrix, float value) {
    DEFAULT_LAUNCH(_memset, matrix, value)
  }

  void download_matrix(const Matrix *src, float *dst) {
    hipblasStatus_t stat = hipblasGetMatrix(src->width, src->height, 
        sizeof(*src->dev_array), 
        src->dev_array, src->width, dst, src->width);
    check(stat != HIPBLAS_STATUS_SUCCESS, "data download failed");
  }

  void print_matrix(Matrix *matrix) {

    // allocate space for matrix on CPU 
    float *array = (float *)malloc(matrix->width * matrix->height *
        sizeof(*matrix->dev_array)); 
    check(!array, "host memory allocation failed"); 

    // copy matrix to CPU
    download_matrix(matrix, array);

    int i, j;
    rng(j, 0, matrix->height) {
      rng(i, 0, matrix->width) {
        printf("%7.0f", array[idx2c(i, j, matrix->width)]);
      }
      printf("\n");
    }

    free(array);
  }

  void free_matrix(Matrix *matrix) {
    free(matrix->dev_array);
  }
}

