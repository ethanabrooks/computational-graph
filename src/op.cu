#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 

#define UN_MAP(name, f_body) \
  __device__ \
  float f_ ## name(float x) { \
    return f_body; \
  } \
  __global__ \
  void _ ## name(int len, float *result, float *a) { \
    SET(result, f_ ## name(a[IDx])) \
  } \
  void map_ ## name(Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name, result, m->dev_array); \
  }

#define BIN_BROADCAST(name, op) \
  __global__ \
  void _ ## name ## _scalar(int len, float *result, float *a, float val) { \
    SET(result, val op a[IDx]) \
  } \
  void broadcast_ ## name(float val, Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar, result, m->dev_array, val); \
  }

#define BIN_BROADCAST_REV(name, op) \
  __global__ \
  void _ ## name ## _scalar_rev(int len, float *result, float *a, float val) { \
    SET(result, a[IDx] op val) \
  } \
  void broadcast_ ## name ## _rev(Matrix *m, float val, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar_rev, result, m->dev_array, val); \
  }

#define BIN_ELEMWISE(name, op) \
  __global__ \
  void _ ## name (int len, float *result, float *a1, float *a2) { \
    SET(result, a1[IDx] op a2[IDx]) \
  } \
  void elemwise_ ## name (Matrix *m1, Matrix *m2, Matrix *result) { \
    check_dims(m1, m2, result); \
    DEFAULT_LAUNCH(_ ## name, result, m1->dev_array, m2->dev_array); \
  }

void check_dims(Matrix *m1, Matrix *m2, Matrix *result) { 
  check(m1->height != m2->height 
     || m1->width  != m2->width
     || m1->height != result->height 
     || m1->width  != result->width, 
      "matrices must have the same dimensions");
}

extern "C" {
  UN_MAP(neg, -x) // map_neg

  BIN_ELEMWISE(mult, *) // elemwise_mult
  BIN_ELEMWISE(add, +) // elemwise_add
  BIN_ELEMWISE(sub, -) // elemwise_sub

  BIN_BROADCAST(mult, *) // broadcast_mult
  BIN_BROADCAST(add, +) // broadcast_add
  BIN_BROADCAST(sub, -) // broadcast_sub

  BIN_BROADCAST_REV(sub, -) // broadcast_sub_rev
}
