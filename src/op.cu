#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "scan.h" 

#define UN_MAP(name, f_body) \
  __device__ \
  float f_ ## name(float x) { \
    return f_body; \
  } \
  __global__ \
  void _ ## name(int len, float *result, const float *a) { \
    SET(result, f_ ## name(a[IDx])) \
  } \
  void map_ ## name(const Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name, result, m->dev_array); \
  }

#define BIN_BROADCAST(name, op) \
  __global__ \
  void _ ## name ## _scalar(int len, float *result, const float *a, float val) { \
    SET(result, val op a[IDx]) \
  } \
  void broadcast_ ## name(float val, const Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar, result, m->dev_array, val); \
  }

#define BIN_BROADCAST_REV(name, op) \
  __global__ \
  void _ ## name ## _scalar_rev(int len, float *result, const float *a, float val) { \
    SET(result, a[IDx] op val) \
  } \
  void broadcast_ ## name ## _rev(const Matrix *m, float val, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar_rev, result, m->dev_array, val); \
  }

#define BIN_ELEMWISE(name, op) \
  __global__ \
  void _ ## name (int len, float *result, const float *a1, const float *a2) { \
    SET(result, a1[IDx] op a2[IDx]) \
  } \
  void elemwise_ ## name (const Matrix *m1, const Matrix *m2, Matrix *result) { \
    check_dims(m1, m2, result); \
    DEFAULT_LAUNCH(_ ## name, result, m1->dev_array, m2->dev_array); \
  }

void check_dims(const Matrix *m1, const Matrix *m2, const Matrix *result) { 
  check(m1->height != m2->height 
     || m1->width  != m2->width
     || m1->height != result->height 
     || m1->width  != result->width, 
      "matrices must have the same dimensions");
}

extern "C" {
  UN_MAP(neg, -x) // map_neg

  BIN_ELEMWISE(mult, *) // elemwise_mult
  BIN_ELEMWISE(add, +) // elemwise_add
  BIN_ELEMWISE(sub, -) // elemwise_sub

  BIN_BROADCAST(mult, *) // broadcast_mult
  BIN_BROADCAST(add, +) // broadcast_add
  BIN_BROADCAST(sub, -) // broadcast_sub

  BIN_BROADCAST_REV(sub, -) // broadcast_sub_rev

  __global__
  void _reduce_equal(int len, const float *a, float x, unsigned int *boolean) {
    printf("%x, TEST TEST TEST\n", boolean);
    if (IDx >= len) return;
    unsigned int equal = a[IDx] == x;
    printf("equal: %d\n", equal);
    atomicAnd(boolean, equal); 
  }

  bool reduce_equal(const Matrix *m, float x) {
    unsigned int *dev_bool;
    hipError_t cudaStat = hipMalloc((void**)&dev_bool,
        sizeof(*dev_bool));
    check(cudaStat != hipSuccess, "hipMalloc failed for `dev_bool` in `reduce_eq`");

    unsigned int t = 1;
    hipMemcpy(dev_bool, &t, sizeof(t), hipMemcpyHostToDevice);

    _reduce_equal<<<blockcount(size(m)), BLOCKSIZE>>>
      (size(m), m->dev_array, x, dev_bool);


    hipMemcpy(&t, &dev_bool, sizeof(t), hipMemcpyDeviceToHost);
    return t == 1;
  }

  float reduce_sum(const Matrix *m) {
    int size_matrix = size(m);
    check(size_matrix == 0, "matrix must have more than 0 elements.");

    // temp buffer stores result of scan
    float *dev_temp;
    hipError_t cudaStat = hipMalloc((void**)&dev_temp,
        size_matrix*sizeof(*dev_temp));
    check(cudaStat != hipSuccess, "hipMalloc failed for `temp` in `reduce_avg`");

    dev_scan(size_matrix, m->dev_array, dev_temp);

    // last element of scan is sum of all but last element of matrix
    float last_scan_val, last_matrix_val;

    hipMemcpy(&last_scan_val, &dev_temp[size_matrix - 1], 
        sizeof(last_scan_val), hipMemcpyDeviceToHost);

    hipMemcpy(&last_matrix_val, &m->dev_array[size_matrix - 1],
        sizeof(last_matrix_val), hipMemcpyDeviceToHost);

    hipFree(dev_temp);
    return last_matrix_val + last_scan_val;
  }
}
