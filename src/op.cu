#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "matrix.h" 
#include "scan.h" 

#define UN_MAP(name, f_body) \
  __device__ \
  float f_ ## name(float x) { \
    return f_body; \
  } \
  __global__ \
  void _ ## name(int len, float *result, float *a) { \
    SET(result, f_ ## name(a[IDx])) \
  } \
  void map_ ## name(Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name, result, m->dev_array); \
  }

#define BIN_BROADCAST(name, op) \
  __global__ \
  void _ ## name ## _scalar(int len, float *result, float *a, float val) { \
    SET(result, val op a[IDx]) \
  } \
  void broadcast_ ## name(float val, Matrix *m, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar, result, m->dev_array, val); \
  }

#define BIN_BROADCAST_REV(name, op) \
  __global__ \
  void _ ## name ## _scalar_rev(int len, float *result, float *a, float val) { \
    SET(result, a[IDx] op val) \
  } \
  void broadcast_ ## name ## _rev(Matrix *m, float val, Matrix *result) { \
    DEFAULT_LAUNCH(_ ## name ## _scalar_rev, result, m->dev_array, val); \
  }

#define BIN_ELEMWISE(name, op) \
  __global__ \
  void _ ## name (int len, float *result, float *a1, float *a2) { \
    SET(result, a1[IDx] op a2[IDx]) \
  } \
  void elemwise_ ## name (Matrix *m1, Matrix *m2, Matrix *result) { \
    check_dims(m1, m2, result); \
    DEFAULT_LAUNCH(_ ## name, result, m1->dev_array, m2->dev_array); \
  }

void check_dims(Matrix *m1, Matrix *m2, Matrix *result) { 
  check(m1->height != m2->height 
     || m1->width  != m2->width
     || m1->height != result->height 
     || m1->width  != result->width, 
      "matrices must have the same dimensions");
}

extern "C" {
  UN_MAP(neg, -x) // map_neg

  BIN_ELEMWISE(mult, *) // elemwise_mult
  BIN_ELEMWISE(add, +) // elemwise_add
  BIN_ELEMWISE(sub, -) // elemwise_sub

  BIN_BROADCAST(mult, *) // broadcast_mult
  BIN_BROADCAST(add, +) // broadcast_add
  BIN_BROADCAST(sub, -) // broadcast_sub

  BIN_BROADCAST_REV(sub, -) // broadcast_sub_rev

  float reduce_sum(Matrix *m) {
    int size_matrix = size(*m);
    check(size_matrix == 0, "matrix must have more than 0 elements.");

    // temp buffer stores result of scan
    float *dev_temp;
    hipError_t cudaStat = hipMalloc((void**)&dev_temp,
        size_matrix*sizeof(*dev_temp));
    check(cudaStat != hipSuccess, "hipMalloc failed for `temp` in `reduce_avg`");

    dev_scan(size_matrix, dev_temp, m->dev_array);

    // last element of scan is sum of all but last element of matrix
    float last_scan_val, last_matrix_val;

    hipMemcpy(&last_scan_val, &dev_temp[size_matrix - 1], 
        sizeof(last_scan_val), hipMemcpyDeviceToHost);

    hipMemcpy(&last_matrix_val, &m->dev_array[size_matrix - 1],
        sizeof(last_matrix_val), hipMemcpyDeviceToHost);

    hipFree(dev_temp);
    return last_matrix_val + last_scan_val;
  }
}
