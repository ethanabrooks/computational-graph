#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include "util.h" 

dim3 blockcount(int count) {
  float numblocks = (count / BLOCKSIZE.x + 1);
  return pow(2, ceil(log2(numblocks)));
}

void check(int condition, const char *msg) {
  if (condition) {
    fprintf(stderr, "ERROR: %s\n", msg);
    exit(EXIT_FAILURE);
  }
}

/*void device2host(float *src, float *dst) {*/
    /*hipError_t cudaStat = hipMemcpy(dev_sum, &z, sizeof(z), hipMemcpyHostToDevice);*/
    /*check(cudaStat != hipSuccess, "hipMemcpy failed");*/
/*}*/
