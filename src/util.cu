#include "hip/hip_runtime.h"
#include <stdio.h> 
#include "util.h" 

dim3 blockcount(int count) {
  float numblocks = (count / BLOCKSIZE.x + 1);
  return pow(2, ceil(log2(numblocks))); \
}

void check(int condition, const char *msg) {
  if (condition) {
    fprintf(stderr, "ERROR: %s\n", msg);
    exit(EXIT_FAILURE);
  }
}
